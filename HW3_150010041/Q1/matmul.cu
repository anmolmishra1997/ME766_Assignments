
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define N 10000

__global__ void arrmul(float * d_A, float * d_B, float * d_C)
{
	long long int idx = blockIdx.x;
	long long int idy = blockIdx.y;
	long long int idz = threadIdx.x;
	long long int index = idz + (idy + idx*gridDim.y) *blockDim.x;
	long long int row = index/N;
	long long int col = index%N;
	// long long int idy = threadIdx.y;

	for(long long int k=0; k<N; k++)
		d_C[index] += d_A[row*N + k] * d_B[k*N + col];

}

void printArray(float* arr)
{
	for (long long int i=0; i<N; i++)
	{
		for (long long int j=0;j<N;j++)
			cout << arr[i*N+j] << " ";
		cout << endl;
	}
	cout << endl;
}

int main(int argc, char** argv)
{

	long long int array_bytes = N*N * sizeof(float);
	float *h_A, *h_B, *h_C;

	h_A = (float*) malloc(N*N * sizeof(float));
	h_B = (float*) malloc(N*N * sizeof(float));
	h_C = (float*) malloc(N*N * sizeof(float));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	srand(time(NULL));

	for(long long int i=0; i<N*N; i++)
	{
		h_A[i] = (double)rand()/(double)RAND_MAX * 1;
		h_B[i] = (double)rand()/(double)RAND_MAX * 1;
		h_C[i] = 0;
	}

	float * d_A;
	float * d_B;
	float * d_C;

	hipMalloc((void **) &d_A, array_bytes);
	hipMalloc((void **) &d_B, array_bytes);
	hipMalloc((void **) &d_C, array_bytes);

	hipEventRecord(start);
	hipMemcpy(d_A, h_A, array_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, array_bytes, hipMemcpyHostToDevice);

	long long int blockx = N/10;
	long long int blocky = N/10;
	long long int threads = 100;

	dim3 block( blockx, blocky );

	arrmul<<<block, threads >>>(d_A, d_B, d_C);

	hipMemcpy(h_C, d_C, array_bytes, hipMemcpyDeviceToHost);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << milliseconds << endl;

	// printArray(h_A);
	// printArray(h_B);
	// printArray(h_C);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}