#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.1415926535

__global__ void mont(float* d_xData, float* d_yData, int* d_count)
{
	int idx = blockIdx.x;
	float rx = d_xData[idx];
	float ry = d_yData[idx];
	if (ry <= sinpi(rx))
		atomicAdd(d_count, 1);

	// float rx = sin(idx*width) + sin((idx+1)*width);
	// atomicAdd(res, c);
}

 
int main(int argc, char** argv)
{
	if (argc != 2)
	{
		printf("Must provide one argument - n\n");
		exit(1);
	}
	int n = atoi(argv[1]);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    hiprandGenerator_t gen;
    float *d_xData, *d_yData;
    // float h_xData[n], h_yData[n];


    hipMalloc((void **)&d_xData, n*sizeof(float));
    hipMalloc((void **)&d_yData, n*sizeof(float));

    int count;

	int* h_count = &count;
	*h_count = 0;

	int* d_count;
	hipMalloc((void **) &d_count, sizeof(int));


    hiprandCreateGenerator(&gen, 
                HIPRAND_RNG_PSEUDO_DEFAULT);
    

    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

	hipEventRecord(start);
    hiprandGenerateUniform(gen, d_xData, n);
    hiprandGenerateUniform(gen, d_yData, n);


    // cudaMemcpy(h_xData, d_xData, n * sizeof(float),
    //     cudaMemcpyDeviceToHost);
    // cudaMemcpy(h_yData, d_yData, n * sizeof(float),
    //     cudaMemcpyDeviceToHost);

	hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);

	mont<<<n, 1 >>>(d_xData, d_yData, d_count);


	hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	float res = PI* (*h_count)/n;

	printf("%f %f \n", res, milliseconds);
}