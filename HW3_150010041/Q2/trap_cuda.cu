
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


#define PI 3.1415926535

__global__ void trap(float width, float* res)
{
	int idx = blockIdx.x;
	float c = sin(idx*width) + sin((idx+1)*width);
	atomicAdd(res, c);
}

int main(int argc, char** argv)
{
	if (argc != 2)
	{
		printf("Must provide one argument - n\n");
		exit(1);
	}

	int n = atoi(argv[1]);
	float width = PI/n;
	float cf = width/2;
	float res;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* h_res = &res;
	*h_res = 0;

	float* d_res;
	hipMalloc((void **) &d_res, sizeof(float));

	hipEventRecord(start);
	hipMemcpy(d_res, h_res, sizeof(float), hipMemcpyHostToDevice);

	trap<<<n, 1 >>>(width, d_res);

	hipMemcpy(h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	res = *h_res * cf;

	printf("%f %f\n", res, milliseconds);

	hipFree(d_res);
}